/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void gpuYoloLayer_x(const float* input, int* num_detections, float* detection_boxes, float* detection_scores,
    int* detection_classes, const float scoreThreshold, const uint netWidth, const uint netHeight,
    const uint numOutputClasses, const uint64_t outputSize, const float* anchors, const int* mask)
{
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (x_id >= outputSize)
    return;

  const float objectness = input[x_id * (5 + numOutputClasses) + 4];

  if (objectness < scoreThreshold)
    return;

  int count = (int)atomicAdd(num_detections, 1);

  float x = (input[x_id * (5 + numOutputClasses) + 0] + anchors[x_id * 2]) * mask[x_id];

  float y = (input[x_id * (5 + numOutputClasses) + 1] + anchors[x_id * 2 + 1]) * mask[x_id];

  float w = __expf(input[x_id * (5 + numOutputClasses) + 2]) * mask[x_id];

  float h = __expf(input[x_id * (5 + numOutputClasses) + 3]) * mask[x_id];

  float maxProb = 0.0f;
  int maxIndex = -1;

  for (uint i = 0; i < numOutputClasses; ++i) {
    float prob = input[x_id * (5 + numOutputClasses) + 5 + i];
    if (prob > maxProb) {
      maxProb = prob;
      maxIndex = i;
    }
  }

  detection_boxes[count * 4 + 0] = x - 0.5 * w;
  detection_boxes[count * 4 + 1] = y - 0.5 * h;
  detection_boxes[count * 4 + 2] = x + 0.5 * w;
  detection_boxes[count * 4 + 3] = y + 0.5 * h;
  detection_scores[count] = objectness * maxProb;
  detection_classes[count] = maxIndex;
}

hipError_t cudaYoloLayer_x(const void* input, void* num_detections, void* detection_boxes, void* detection_scores,
    void* detection_classes, const uint& batchSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& numOutputClasses, const void* anchors, const void* mask, hipStream_t stream);

hipError_t cudaYoloLayer_x(const void* input, void* num_detections, void* detection_boxes, void* detection_scores,
    void* detection_classes, const uint& batchSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& numOutputClasses, const void* anchors, const void* mask, hipStream_t stream)
{
  int threads_per_block = 16;
  int number_of_blocks = (outputSize / threads_per_block) + 1;

  for (unsigned int batch = 0; batch < batchSize; ++batch) {
    gpuYoloLayer_x<<<number_of_blocks, threads_per_block, 0, stream>>>(
        reinterpret_cast<const float*>(input) + (batch * (5 + numOutputClasses) * outputSize),
        reinterpret_cast<int*>(num_detections) + (batch),
        reinterpret_cast<float*>(detection_boxes) + (batch * 4 * outputSize),
        reinterpret_cast<float*>(detection_scores) + (batch * outputSize),
        reinterpret_cast<int*>(detection_classes) + (batch * outputSize),
        scoreThreshold, netWidth, netHeight, numOutputClasses, outputSize, reinterpret_cast<const float*>(anchors),
        reinterpret_cast<const int*>(mask));
  }
  return hipGetLastError();
}
