/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void gpuYoloLayer_v8(const float* input, int* num_detections, float* detection_boxes, float* detection_scores,
    int* detection_classes, const float scoreThreshold, const uint netWidth, const uint netHeight,
    const uint numOutputClasses, const uint64_t outputSize)
{
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (x_id >= outputSize)
    return;

  float maxProb = 0.0f;
  int maxIndex = -1;

  for (uint i = 0; i < numOutputClasses; ++i) {
    float prob = input[x_id * (4 + numOutputClasses) + 4 + i];
    if (prob > maxProb) {
      maxProb = prob;
      maxIndex = i;
    }
  }

  if (maxProb < scoreThreshold)
    return;

  int count = (int)atomicAdd(num_detections, 1);

  detection_boxes[count * 4 + 0] = input[x_id * (4 + numOutputClasses) + 0];
  detection_boxes[count * 4 + 1] = input[x_id * (4 + numOutputClasses) + 1];
  detection_boxes[count * 4 + 2] = input[x_id * (4 + numOutputClasses) + 2];
  detection_boxes[count * 4 + 3] = input[x_id * (4 + numOutputClasses) + 3];
  detection_scores[count] = maxProb;
  detection_classes[count] = maxIndex;
}

hipError_t cudaYoloLayer_v8(const void* input, void* num_detections, void* detection_boxes, void* detection_scores,
    void* detection_classes, const uint& batchSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& numOutputClasses, hipStream_t stream);

hipError_t cudaYoloLayer_v8(const void* input, void* num_detections, void* detection_boxes, void* detection_scores,
    void* detection_classes, const uint& batchSize, uint64_t& outputSize, const float& scoreThreshold, const uint& netWidth,
    const uint& netHeight, const uint& numOutputClasses, hipStream_t stream)
{
  int threads_per_block = 16;
  int number_of_blocks = (outputSize / threads_per_block) + 1;

  for (unsigned int batch = 0; batch < batchSize; ++batch) {
    gpuYoloLayer_v8<<<number_of_blocks, threads_per_block, 0, stream>>>(
        reinterpret_cast<const float*>(input) + (batch * (4 + numOutputClasses) * outputSize),
        reinterpret_cast<int*>(num_detections) + (batch),
        reinterpret_cast<float*>(detection_boxes) + (batch * 4 * outputSize),
        reinterpret_cast<float*>(detection_scores) + (batch * outputSize),
        reinterpret_cast<int*>(detection_classes) + (batch * outputSize),
        scoreThreshold, netWidth, netHeight, numOutputClasses, outputSize);
  }
  return hipGetLastError();
}
